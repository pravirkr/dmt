#include <benchmark/benchmark.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>

#include <dmt/fdmt_gpu.hpp>

#define BENCH_CUDA_TRY(call)                                                         \
  do {                                                                               \
    auto const status = (call);                                                      \
    if (hipSuccess != status) { throw std::runtime_error("CUDA error detected."); } \
  } while (0);


class cuda_event_timer {
 public:
  /**
   * @brief Constructs a `cuda_event_timer` beginning a manual timing range.
   *
   * Optionally flushes L2 cache.
   *
   * @param[in,out] state  This is the benchmark::State whose timer we are going
   * to update.
   * @param[in] flush_l2_cache_ whether or not to flush the L2 cache before
   *                            every iteration.
   * @param[in] stream_ The CUDA stream we are measuring time on.
   */
  cuda_event_timer(benchmark::State &state, bool flush_l2_cache = false, hipStream_t stream = 0)
    : p_state(&state), stream_(stream)
  {
    // flush all of L2$
    if (flush_l2_cache) {
      int current_device = 0;
      BENCH_CUDA_TRY(hipGetDevice(&current_device));

      int l2_cache_bytes = 0;
      BENCH_CUDA_TRY(
        hipDeviceGetAttribute(&l2_cache_bytes, hipDeviceAttributeL2CacheSize, current_device));

      if (l2_cache_bytes > 0) {
        const int memset_value = 0;
        int *l2_cache_buffer   = nullptr;
        BENCH_CUDA_TRY(hipMalloc(&l2_cache_buffer, l2_cache_bytes));
        BENCH_CUDA_TRY(hipMemsetAsync(l2_cache_buffer, memset_value, l2_cache_bytes, stream_));
        BENCH_CUDA_TRY(hipFree(l2_cache_buffer));
      }
    }

    BENCH_CUDA_TRY(hipEventCreate(&start_));
    BENCH_CUDA_TRY(hipEventCreate(&stop_));
    BENCH_CUDA_TRY(hipEventRecord(start_, stream_));
  }

  cuda_event_timer() = delete;

  /**
   * @brief Destroy the `cuda_event_timer` and ending the manual time range.
   *
   */
  ~cuda_event_timer()
  {
    BENCH_CUDA_TRY(hipEventRecord(stop_, stream_));
    BENCH_CUDA_TRY(hipEventSynchronize(stop_));
    float milliseconds = 0.0f;
    BENCH_CUDA_TRY(hipEventElapsedTime(&milliseconds, start_, stop_));
    p_state->SetIterationTime(milliseconds / (1000.0f));
    BENCH_CUDA_TRY(hipEventDestroy(start_));
    BENCH_CUDA_TRY(hipEventDestroy(stop_));
  }

 private:
  hipEvent_t start_;
  hipEvent_t stop_;
  hipStream_t stream_;
  benchmark::State *p_state;
};

class FDMTGPUFixture : public benchmark::Fixture {
public:
    void SetUp(const ::benchmark::State& state) override {
        f_min  = 704.0F;
        f_max  = 1216.0F;
        nchans = 4096;
        tsamp  = 0.00008192F;
        dt_max = 2048;
        nsamps = state.range(0);
    }

    void TearDown(const ::benchmark::State& /*unused*/) override {}

    template <typename T>
    thrust::device_vector<T> generate_vector_gpu(size_t size) {
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<T> dist(0.0, 1.0);

        thrust::device_vector<T> vec(size);
        thrust::transform(thrust::counting_iterator<size_t>(0),
                          thrust::counting_iterator<size_t>(size), vec.begin(),
                          [=] __device__(size_t idx) mutable { return dist(rng); });

        return vec;
    }

    float f_min{};
    float f_max{};
    size_t nchans{};
    float tsamp{};
    size_t dt_max{};
    size_t nsamps{};
};

BENCHMARK_DEFINE_F(FDMTGPUFixture, BM_fdmt_plan_gpu)(benchmark::State& state) {
    for (auto _ : state) {
        cuda_event_timer raii{state};
        FDMTGPU fdmt(f_min, f_max, nchans, nsamps, tsamp, dt_max);
    }
}

BENCHMARK_DEFINE_F(FDMTGPUFixture, BM_fdmt_initialise_gpu)
(benchmark::State& state) {
    FDMTGPU fdmt(f_min, f_max, nchans, nsamps, tsamp, dt_max);
    auto waterfall_d       = generate_vector_gpu<float>(nchans * nsamps);
    float* waterfall_d_ptr = thrust::raw_pointer_cast(waterfall_d.data());
    const auto& plan       = fdmt.get_plan();
    const auto state_size  = plan.state_shape[0][3] * plan.state_shape[0][4];
    thrust::device_vector<float> state_init_d(state_size, 0.0F);
    float* state_init_d_ptr = thrust::raw_pointer_cast(state_init_d.data());
    for (auto _ : state) {
        cuda_event_timer raii{state};
        fdmt.initialise(waterfall_d_ptr, state_init_d_ptr);
    }
}

BENCHMARK_DEFINE_F(FDMTGPUFixture, BM_fdmt_execute_gpu)
(benchmark::State& state) {
    FDMTGPU fdmt(f_min, f_max, nchans, nsamps, tsamp, dt_max);
    auto waterfall_d       = generate_vector_gpu<float>(nchans * nsamps);
    float* waterfall_d_ptr = thrust::raw_pointer_cast(waterfall_d.data());
    thrust::device_vector<float> dmt_d(fdmt.get_dt_grid_final().size() * nsamps,
                                       0.0F);
    float* dmt_d_ptr = thrust::raw_pointer_cast(dmt_d.data());
    for (auto _ : state) {
        cuda_event_timer raii{state};
        fdmt.execute(waterfall_d_ptr, waterfall_d.size(), dmt_d_ptr,
                     dmt_d.size());
    }
}

BENCHMARK_DEFINE_F(FDMTGPUFixture, BM_fdmt_overall_gpu)
(benchmark::State& state) {
    auto waterfall_d       = generate_vector_gpu<float>(nchans * nsamps);
    float* waterfall_d_ptr = thrust::raw_pointer_cast(waterfall_d.data());

    for (auto _ : state) {
        cuda_event_timer raii{state};
        FDMTGPU fdmt(f_min, f_max, nchans, nsamps, tsamp, dt_max);
        state.PauseTiming();
        thrust::device_vector<float> dmt_d(
            fdmt.get_dt_grid_final().size() * nsamps, 0.0F);
        float* dmt_d_ptr = thrust::raw_pointer_cast(dmt_d.data());
        state.ResumeTiming();
        fdmt.execute(waterfall_d_ptr, waterfall_d.size(), dmt_d_ptr,
                     dmt_d.size());
    }
}

constexpr size_t kMinNsamps = 1 << 11;
constexpr size_t kMaxNsamps = 1 << 15;

BENCHMARK_REGISTER_F(FDMTGPUFixture, BM_fdmt_plan_gpu)
    ->RangeMultiplier(2)
    ->Range(kMinNsamps, kMaxNsamps)
    ->UseManualTime();
BENCHMARK_REGISTER_F(FDMTGPUFixture, BM_fdmt_initialise_gpu)
    ->RangeMultiplier(2)
    ->Range(kMinNsamps, kMaxNsamps)
    ->UseManualTime();
BENCHMARK_REGISTER_F(FDMTGPUFixture, BM_fdmt_execute_gpu)
    ->RangeMultiplier(2)
    ->Range(kMinNsamps, kMaxNsamps)
    ->UseManualTime();
BENCHMARK_REGISTER_F(FDMTGPUFixture, BM_fdmt_overall_gpu)
    ->RangeMultiplier(2)
    ->Range(kMinNsamps, kMaxNsamps)
    ->UseManualTime();

//BENCHMARK_MAIN();
